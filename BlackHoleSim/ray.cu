#include "hip/hip_runtime.h"
#include "ray.h"

//__host__ __device__ bool ray::hit_sphere(sphere s) {
//	vec3 orig_diff = (orig - s.get_origin());
//	float a = norm(dir) * orig_diff;
//	float b = orig_diff * orig_diff - s.get_radius_sqr() * s.get_radius_sqr();
//	return (a * a - b)>=0;
//}

__device__ bool ray::hit_sphere(sphere s) {
	vec3_t orig_diff = (orig - s.get_origin());
	float a = norm(dir) * orig_diff;
	float b = orig_diff * orig_diff - s.get_radius_sqr() * s.get_radius_sqr();
	return (a * a - b) >= 0;
}

__device__ vec3_t ray::march(sphere** obj_ls, sphere blackhole, int count) {
	vec3_t next_orig;
	vec3_t color = hdr(v, u);
	vec3_t t = cross(dir, norm(blackhole.get_origin() - orig));
	vec3_t k = norm(t);
	for (int i = 0; i < n_seg; i++) {
		next_orig = orig + delta * dir;
		t = cross(dir, norm(blackhole.get_origin() - orig));

		for (int j = 0; j < count; j++) {
			if (obj_ls[j]->is_inside(next_orig, color)) {
				return color;
			}
		}
		if (blackhole.is_inside(next_orig, color)) {
			return color;
		}
		dir = rotate(dir, k, blackhole.get_deflection(next_orig, 0.01f) * (t * t));
		orig = next_orig;
	}
	color = hdr(((dir.y + 1) / 2 * 256), ((2 - dir.x) / 2 * 512));
	return color;
}

__device__ vec3_t ray::march(sphere_t* obj_ls, sphere_t* blackhole, int count, disk_t* disk) {
	vec3_t next_orig;
	vec3_t color;
	vec3_t t = cross(dir, norm(blackhole->position - orig));
	vec3_t k = norm(t);
	for (int i = 0; i < n_seg; i++) {
		next_orig = orig + DELTA* dir;
		t = cross(dir, norm(blackhole->position - orig));
		if (is_inside(*blackhole, next_orig, color)) {
			return color;
		}
		else if (hit_disk(*disk, orig, dir, color)) {
			return color;
		}

		for (int j = 0; j < count; j++) {
			if (is_inside(obj_ls[j], next_orig, color)) {
				return color;
			}
		}


		dir = norm(rotate(dir, k, get_deflection(*blackhole, next_orig) * (t * t)));
		orig = next_orig;
	}
	color = hdr(((dir.y + 1) / 2 * 256), ((2 - dir.x) / 2 * 512));
	return color;
	}

__device__ vec3_t march(ray_t* r, cv::cuda::PtrStepSz<vec3_t> hdr, sphere_t* obj_ls, sphere_t* blackhole, int count, disk_t* disk) {
	//vec3_t next_orig;
	vec3_t color = { -1,-1,-1 };
	vec3_t r_g = blackhole->position - r->orig;
	float d = r_g * r_g;
	/*vec3_t t = cross(r->dir, norm(blackhole->position - r->orig));
	vec3_t k = norm(t);
	*/

	bool done = false;
	#pragma unroll 4
	for (int i = 0; i < N_STEPS; i++) {

		r->orig = r->orig + DELTA * r->dir;
		r_g = blackhole->position - r->orig;
		d = r_g * r_g;
		//t = cross(r->dir, norm(blackhole->position - r->orig));
		done|=is_inside(*blackhole, r_g, r->orig, color);
		done|=hit_disk(*disk, r, color);
		/*if (is_inside(*blackhole, r_g, r->orig, color)) {
			return color;
		}

		if (hit_disk(*disk, r, color)) {
			return color;
		}*/

		for (int j = 0; j < count; j++) {
			/*if (is_inside(obj_ls[j], r->orig, color)) {
				return color;
			}*/
			done|=is_inside(obj_ls[j], r->orig, color);
		}

		d = (3.0f / 2.0f) * 0.00005f / (d * d * d);
		r->dir = norm(r->dir + d * r_g);

		if (done) {
			break;
		}
	}
	if(color.x<0)
		color = hdr((((asinf((r->dir.y)) + (PI / 2)) / PI)) * 512, (1 - (atan2f(r->dir.z, r->dir.x) + PI) / (2 * PI)) * 1024);
	return color;
}

__device__ vec3_t march(ray_t& r, cv::cuda::PtrStepSz<vec3_t> hdr, sphere_t* obj_ls, sphere_t* blackhole, int count, disk_t* disk) {
	//vec3_t next_orig;
	vec3_t color;
	//vec3_t t = cross(r.dir, norm(blackhole->position - r.orig));
	vec3_t r_g = blackhole->position - r.orig;
	float d = r_g * r_g;
	//vec3_t k = norm(t);

	for (int i = 0; i < N_STEPS; i++) {

		r.orig = r.orig + DELTA * r.dir;
		r_g = blackhole->position - r.orig;
		d = r_g * r_g;

		//t = cross(r.dir, norm(blackhole->position - r.orig));

		if (is_inside(*blackhole, r_g, r.orig, color)) {
			return color;
		}

		if (hit_disk(*disk, r.orig, r.dir, color)) {
			return color;
		}

		for (int j = 0; j < count; j++) {
			if (is_inside(obj_ls[j], r.orig, color)) {
				return color;
			}
		}

		r.dir = norm(r.dir + (3.0f / 2.0f) * 0.00005f * r_g / (d * d * d));
		//r.dir = norm(rotate(r.dir, k, get_deflection(*blackhole, next_orig) * (t * t)));
		//r.orig = next_orig; 
	}
	color = hdr((((asinf((r.dir.y)) + (PI / 2)) / PI)) * 512, (1 - (atan2f(r.dir.z, r.dir.x) + PI) / (2 * PI)) * 1024);
	return color;
}
