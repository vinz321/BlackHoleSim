#include "hip/hip_runtime.h"
#include "ray.h"

//SHARED
__device__ vec3_t march(ray_t* r, cv::cuda::PtrStepSz<vec3_t> hdr, sphere_t* obj_ls, sphere_t* blackhole, int count, disk_t* disk) {
	vec3_t color = { -1,-1,-1 };
	vec3_t r_g = blackhole->position - r->orig;
	float d = r_g * r_g;
	bool done = false;
	#pragma unroll 4
	for (int i = 0; i < N_STEPS; i++) {

		r->orig = r->orig + DELTA * r->dir;
		r_g = blackhole->position - r->orig;
		d = r_g * r_g;

		done|=is_inside(*blackhole, r_g, r->orig, color);
		done|=hit_disk(*disk, r, color);

		for (int j = 0; j < count; j++) {
			done|=is_inside(obj_ls[j], r->orig, color);
		}

		d = (3.0f / 2.0f) * 0.00005f / (d * d * d);
		r->dir = norm(r->dir + d * r_g);

		if (done) {
			break;
		}
	}
	if(color.x<0)
		color = hdr((((asinf((r->dir.y)) + (PI / 2)) / PI)) * 512, (1 - (atan2f(r->dir.z, r->dir.x) + PI) / (2 * PI)) * 1024);
	return color;
}

//BASELINE & CONSTANT
__device__ vec3_t march(ray_t& r, cv::cuda::PtrStepSz<vec3_t> hdr, sphere_t* obj_ls, sphere_t* blackhole, int count, disk_t* disk) {
	vec3_t color;
	vec3_t r_g = blackhole->position - r.orig;
	float d = r_g * r_g;

	for (int i = 0; i < N_STEPS; i++) {

		r.orig = r.orig + DELTA * r.dir;
		r_g = blackhole->position - r.orig;
		d = r_g * r_g;

		if (is_inside(*blackhole, r_g, r.orig, color)) {
			return color;
		}

		if (hit_disk(*disk, r.orig, r.dir, color)) {
			return color;
		}

		for (int j = 0; j < count; j++) {
			if (is_inside(obj_ls[j], r.orig, color)) {
				return color;
			}
		}
		r.dir = norm(r.dir + (3.0f / 2.0f) * 0.00005f * r_g / (d * d * d));
	}
	color = hdr((((asinf((r.dir.y)) + (PI / 2)) / PI)) * 512, (1 - (atan2f(r.dir.z, r.dir.x) + PI) / (2 * PI)) * 1024);
	return color;
}
