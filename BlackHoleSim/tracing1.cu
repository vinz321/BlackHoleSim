#include "hip/hip_runtime.h"
#include "tracing.h"



__device__ vec3 px_color(ray& r) {
    if (r.hit_sphere(sphere(vec3{ 0.0f, 0.0f, -1.0f }, 0.5)))
        return vec3{ 1.0f, 0.0f, 0.0f };
    vec3 unit_direction = norm(r.get_dir());
    float t = 0.5f * (unit_direction.y + 1.0f);
    return (1.0f - t) * vec3 { 1.0, 1.0, 1.0 } + t * vec3{ 0.5, 0.7, 1.0 };
}

__global__ void render(vec3* img, int max_x, int max_y, camera* cam) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    cam = new camera(vec3{ 0.0f, 0.0f, -1.0f },
        vec3{ 0.0f, 0.0f, 1.0f },
        vec3{ 0.0f, 1.0f, 0.0f },
        60,
        max_x / max_y);

    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    ray r = cam->get_ray(u, v);
    img[pixel_index] = px_color(r);
}

