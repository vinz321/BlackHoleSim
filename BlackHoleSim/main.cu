#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/opencv.hpp>
#include "tracing.h"
#include <math.h>
#include "sky_engine.h"

//#include "cudaKern.h"

using namespace cv::cuda;

__global__ void test_kern() {
	vec3_t test = { 1,2,3 };
	vec3_t test2 = test + test;
}

int main() {
	hipProfilerStart();
	vec3_t test = { 1,2,3 };
	test = test + test;

	std::cout << sizeof(object)<< std::endl;

	std::cout << sizeof(sphere) << std::endl;

	std::cout << sizeof(object*) << std::endl;
	float img_w = 512;
	float img_h = 256;
	float angle = 0;
	vec3_t cam_pos= vec3_t{ 0,2*sinf(angle),-2*cosf(angle)};
	vec3_t cam_dir= vec3_t{ 0,-sinf(angle),cosf(angle)};

	Mat3f hdr = hdriread(img_w, img_h);

	size_t free, total;
	hipMemGetInfo(&free, &total);
	
	std::cout << "Free: " << free << " Total: " << total << std::endl;

	cam_pos = vec3_t{ 0, 0, 0 };
	cam_dir = vec3_t{ 0, 0, 1 };

	int nDevices;
	hipGetDeviceCount(&nDevices);

	printf("Number of devices: %d\n", nDevices);

	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (MHz): %d\n",
			prop.memoryClockRate / 1024);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
			2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
		printf("  Total global memory (Gbytes) %.1f\n", (float)(prop.totalGlobalMem) / 1024.0 / 1024.0 / 1024.0);
		printf("  Shared memory per block (Kbytes) %.1f\n", (float)(prop.sharedMemPerBlock) / 1024.0);
		printf("  minor-major: %d-%d\n", prop.minor, prop.major);
		printf("  Warp-size: %d\n", prop.warpSize);
		printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
		printf("  Concurrent computation/communication: %s\n\n", prop.deviceOverlap ? "yes" : "no");
	}

	while (true)
	{
		sphere_t* scene = createSceneStruct(2*angle);
		cam_pos = vec3_t{ 2*cosf(angle), 2* sinf(angle), -0.5f};
		//cam_dir = vec3_t{ -sinf(angle),-sinf(PI / 2 * 0.95f)*cosf(angle),cosf(PI / 2 * 0.95f)};

		cam_dir = norm(vec3_t{0,0,0} - cam_pos);
		camera_t cam = make_cam(cam_pos, cam_dir, vec3_t{ 0,0,1}, 120, (float)img_w / img_h);
		cv::Mat m = renderScene(img_w, img_h, &cam, angle, hdr, scene, (disk_t*)(scene + 3));
		
		cv::imshow("Output", m);
		angle += 0.1f;
		//hipMemGetInfo(&free, &total);
		//std::cout << "Free: " << free << " Total: " << total << std::endl;
		if ((cv::waitKey(1) & 0xFF) == 'q') {
			//hipProfilerStop();
			break;
		}
	}

	return 0;
	
}