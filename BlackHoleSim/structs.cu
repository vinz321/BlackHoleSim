#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "structs.h"

__host__ __device__ vec3_t operator+ (const vec3_t& x, const vec3_t& y) {
	float a = x.x + y.x;
	float b = x.y + y.y;
	float c = x.z + y.z;
	return vec3_t{ a,b,c };

	//return vec3_t{ x.x + y.x,
	//	x.y + y.y,

	//	x.z + y.z };
}

__host__ __device__ float operator* (const vec3_t& x, const vec3_t& y) {
	float a = x.x * y.x;
	a = a+ x.y * y.y;
	a = a + x.z * y.z;

	return a;
	//return x.x * y.x +
	//	x.y * y.y +
	//	x.z * y.z;
}
__host__ __device__ vec3_t operator* (const float& y, const vec3_t& x) {
	return vec3_t{ x.x * y,x.y * y, x.z * y };
}

__host__ __device__ vec3_t operator- (const vec3_t& x, const vec3_t& y) {
	float a = x.x - y.x;
	float b = x.y - y.y;
	float c = x.z - y.z;

	return vec3_t{ a,b,c };
	/*return vec3_t{ x.x ,
		x.y - y.y,
		x.z - y.z };*/
}

__host__ __device__ vec3_t operator/ (const vec3_t& x, const float& y) {
	return vec3_t{ x.x / y,
		x.y / y,
		x.z / y };
}
__host__ __device__ vec3_t norm(const vec3_t& v) {
	return v / sqrtf(v * v);
}
__host__ __device__ vec3_t cross(const vec3_t& x, const vec3_t& y) {
	float a=x.y* y.z - x.z * y.y;
	float b = -(x.x * y.z - x.z * y.x);
	float c = x.x * y.y - x.y * y.x;
	
	return vec3_t{ a,b,c };
	//return vec3_t{ x.y * y.z - x.z * y.y, -(x.x * y.z - x.z * y.x), x.x * y.y - x.y * y.x };
}

__host__ __device__ vec3_t rotate(const vec3_t& x, const vec3_t& k, float theta) {
	float cos = cosf(theta);
	float sin = sinf(theta);
	vec3_t a = cos * x;
	a = a+ sin * cross(k, x);
	float b = (k * x) * (1 - cos);
	vec3_t c = b * k;
	
	return a+c;
	//return cos * x + sin * cross(k, x) + (k * x) * (1 - cos) * k;
}

__host__ __device__ bool hit_disk(disk_t& disk, vec3_t& point, vec3_t& dir, float step_size, vec3_t& color) {
	float dot = disk.normal * dir; //Dot proudct normal and ray direction
	vec3_t rel_pos = (point - disk.position); //relative position point to disk
	float d = rel_pos * disk.normal; //distance point - diskplane

	if (d * dot > 0 || d > step_size || d < -step_size) {
		return false;
	}

	vec3_t plane_pos = rel_pos - d * disk.normal;
	float r = sqrt(plane_pos * plane_pos);

	if (r<disk.radius1 || r>disk.radius2) {
		return false;
	}
	color = disk.color;
	return true;
}