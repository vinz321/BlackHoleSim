#include "hip/hip_runtime.h"
#include "cudaKern.h"

using namespace std;
using namespace cv;
using namespace cv::cuda;

__device__ vec3 get(vec3* unrolled_matrix, uint x, uint y, uint z, uint size) {
	unsigned long idx = x + y * size + z * size * size;

	return unrolled_matrix[idx];
}

__device__ void set(vec3* unrolled_matrix, vec3 value, uint x, uint y, uint z, uint size) {
	unsigned long idx = x + y * size + z * size * size;
	unrolled_matrix[idx] = value;
}

__global__ void _gravity_field(PtrStepSz<vec3> output) {
	//int bidx = blockIdx.x + blockIdx.y * gridDim.x ;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//int idx = x + y * gridDim.x * blockDim.x;

	vec3 t = { x / 256.0f,y / 256.0f,0 };

	output(y, x) = t;
}

__global__ void test_kern(vec3* unrolled) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	vec3 t = {
		x / 256.0f,
		y / 256.0f,
		0
	};
	set(unrolled, t, x, y, 0, 256);
}

Mat calc_gravity_field() {
	vec3* values;
	vec3* values_gpu;
	hipMalloc(&values_gpu, 256 * 256 * 3 * sizeof(vec3));
	hipHostMalloc(&values, 256 * 256 * 3 * sizeof(vec3));
	hipMemset(values_gpu, 0, 256 * 256 * 3 * sizeof(vec3));


	Mat3f test(256, 256);
	test.setTo(Vec3f(0, .5f, 1));

	GpuMat t_gpu;
	t_gpu.upload(test);

	dim3 grid_size(8, 8);
	dim3 block_size(32, 32);
	_gravity_field <<<grid_size, block_size >>> (t_gpu);
	//test_kern<<<grid_size,block_size>>>(values_gpu);
	//hipError_t err = hipGetLastError();
	//printf("%s", hipGetErrorString(err));

	//hipMemcpy(values, values_gpu, 256 * 256 * 3 * sizeof(gpu_vec3), hipMemcpyDeviceToHost);
	t_gpu.download(test);
	//hipDeviceSynchronize();

	//test = Mat3f();

	return test;
}

Mat renderize() {
	int x = 256;
	int y = 256;
	camera* cam;
	Mat3f frame(x, y);
	GpuMat t_gpu;
	
	frame.setTo(Vec3f(0,0,0));
	t_gpu.upload(frame);

	dim3 grid_size(8, 8);
	dim3 block_size(32, 32);

	render << <grid_size, block_size >> > (t_gpu, x, y, cam);
	hipDeviceSynchronize();
	printf(hipGetErrorString(hipGetLastError()));

	t_gpu.download(frame);
	//std::cout << frame << std::endl;

	return frame;
}

int add_bi(int a, int b) {
	return a + b;
}