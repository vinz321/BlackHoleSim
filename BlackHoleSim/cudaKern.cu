#include "hip/hip_runtime.h"

#include "cudaKern.h"

using namespace cv;
using namespace cv::cuda;

__device__ gpu_vec3 get(gpu_vec3 *unrolled_matrix,uint x,uint y,uint z, uint size) {
	unsigned long idx = x + y * size + z * size * size;

	return unrolled_matrix[idx];
}

__device__ void set(gpu_vec3* unrolled_matrix, gpu_vec3 value, uint x, uint y, uint z, uint size) {
	unsigned long idx = x + y * size + z * size * size;
	unrolled_matrix[idx]=value;
}

__global__ void _gravity_field(PtrStepSz<gpu_vec3> output) {
	//int bidx = blockIdx.x + blockIdx.y * gridDim.x ;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//int idx = x + y * gridDim.x * blockDim.x;

	gpu_vec3 t = { x / 256.0f,y / 256.0f,0 };

	output(y, x) = t;
}

__global__ void test_kern(gpu_vec3 *unrolled) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	gpu_vec3 t = {
		x / 256.0f,
		y / 256.0f,
		0
	};
	set(unrolled, t, x, y, 0, 256);
}

Mat calc_gravity_field() {
	gpu_vec3* values;
	gpu_vec3* values_gpu;
	hipMalloc(&values_gpu	, 256 * 256 * 3 * sizeof(gpu_vec3));
	hipHostMalloc(&values, 256 * 256 * 3 * sizeof(gpu_vec3));
	hipMemset(values_gpu, 0, 256 * 256 * 3 * sizeof(gpu_vec3));


	Mat3f test(256,256);
	test.setTo(Vec3f(0, .5f, 1));


	
	GpuMat t_gpu;
	t_gpu.upload(test);

	dim3 grid_size(8,8);
	dim3 block_size(32,32);
	_gravity_field <<<grid_size,block_size>>> (t_gpu);
	//test_kern<<<grid_size,block_size>>>(values_gpu);
	//hipError_t err = hipGetLastError();
	//printf("%s", hipGetErrorString(err));

	//hipMemcpy(values, values_gpu, 256 * 256 * 3 * sizeof(gpu_vec3), hipMemcpyDeviceToHost);
	t_gpu.download(test);
	//hipDeviceSynchronize();

	//test = Mat3f();

	return test;
}

int add_bi(int a, int b) {
	return a + b;
}