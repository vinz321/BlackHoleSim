#include "hip/hip_runtime.h"
#pragma once

#include "tracing.h"
#include "sky_engine.h"
#include <iostream>

using namespace std;

__device__ vec3_t color(ray r) {
    if (r.hit_sphere(sphere(vec3_t{ 0.0f, 0.0f, 0.0f }, 0.5)))
        return vec3_t{ 0.0f, 0.0f, 0.0f };
    //else return HDRI color correspondent to this ray
    vec3_t unit_direction = norm(r.get_dir());
    
    float t = 0.5f * (unit_direction.y + 1.0f);
    return (1.0f - t) * vec3_t{1.0, 1.0, 1.0} + t * vec3_t{ 0.5, 0.7, 1.0 };
}

__constant__ sphere_t test_const[8];
__constant__ camera_t cam_const[1];

__global__ void render_base(cv::cuda::PtrStepSz<float3> img, cv::cuda::PtrStepSz<vec3_t> hdr, int max_x, int max_y, camera_t* cam_o, sphere_t* ls, int count, disk_t* disk_s) {
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    //sphere s = sphere(*ls[0]);
    if ((i >= max_x) || (j >= max_y)) return;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    
    ray_t r1 = ray_t{ cam_o->origin, cam_o->lower_left_corner + (u * cam_o->horizontal) + (v * cam_o->vertical) - cam_o->origin };
    vec3_t col = march(r1, hdr, ls + 1, ls, count, disk_s);
    img(j, i) = reinterpret_cast<float3 *> (&col)[0];
}

__global__ void render_shared(cv::cuda::PtrStepSz<vec3_t> img, cv::cuda::PtrStepSz<vec3_t> hdr, int max_x, int max_y, int count) {
    
    __shared__ ray_t rays[128];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    int rid = threadIdx.x + threadIdx.y * blockDim.x;
   
    if ((i >= max_x) || (j >= max_y)) return;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);

    rays[rid] = ray_t{cam_const->origin, cam_const->lower_left_corner + (u * cam_const->horizontal) + (v * cam_const->vertical) - cam_const->origin };
    
    //printf("Thread: %d %d: %p %p %p,  %p %p %p\n", i, j, &(rays[rid].orig.x), &(rays[rid].orig.y), &(rays[rid].orig.z), &(rays[rid].dir.x), &(rays[rid].dir.y), &(rays[rid].dir.z));
    //vec3_t col = 
    img(j, i) = march(rays + rid, hdr, test_const + 1, test_const, count, (disk_t*)(test_const + count + 1));
}

__global__ void render_constant(cv::cuda::PtrStepSz<vec3_t> img, cv::cuda::PtrStepSz<vec3_t> hdr, int max_x, int max_y, int count) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    //sphere s = sphere(*ls[0]);
    if ((i >= max_x) || (j >= max_y)) return;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);

    //ray r = ray(cam->origin, cam->lower_left_corner + (u * cam->horizontal) + (v * cam->vertical) - cam->origin, i, j, hdr);
    ray_t r1 = ray_t{ cam_const->origin, cam_const->lower_left_corner + (u * cam_const->horizontal) + (v * cam_const->vertical) - cam_const->origin };

    //vec3_t col = r.march(ls, blackhole, count, disk);
    vec3_t col = march(r1, hdr, test_const+1, test_const, count, (disk_t *)(test_const + 1 + count));

    //__syncthreads();
    img(j, i) = col;
}

__global__ void instantiate_scene(sphere ** ls, int count) {
    for (int i = 0; i < count; i++) {
        ls[i] = new sphere(*(ls[i]));
    }
}


sphere_t* createSceneStruct(float angle, hipStream_t stream) {
    int size = 2;

    sphere_t* scene;
    hipHostMalloc(&scene, sizeof(sphere_t) * (size+1)+sizeof(disk_t));
    sphere_t* scene_gpu;
    hipMalloc(&scene_gpu, sizeof(sphere_t) * (size+1)+sizeof(disk_t));

    scene[0] = sphere_t{ { 0,0,0 }, 0.2f, { 0,0,0 } , 0.0025f };
    scene[1] = sphere_t{ vec3_t{ -.8f * cosf(angle) , .8f * sinf(angle), 0 }, 0.1f, { 1,1,.8f }, 0 };
    scene[2] = sphere_t{ vec3_t{ 0.95f,0,0 }, 0.05f, {.9f ,1, 1 }, 0 };

    *(disk_t*)(scene + 3) = disk_t{ {0,0,0}, 0.25f, 0.6f, {1,1,1} ,{0,0,1} };

    hipMemcpyAsync(scene_gpu, scene, sizeof(sphere_t) * (size + 1) + sizeof(disk_t), hipMemcpyHostToDevice);

    return scene_gpu;
}

void createSceneInConstant(float angle, hipStream_t stream, camera_t *cam) {
    int size = 3;

    sphere_t* scene;
    hipHostMalloc(&scene, sizeof(sphere_t) * size + sizeof(disk_t));

    scene[0] = sphere_t{ { 0,0,0 }, 0.2f, { 0,0,0 } , 0.0025f };
    scene[1] = sphere_t{ vec3_t{ -.8f * cosf(angle) , .8f * sinf(angle), 0 }, 0.1f, { 1,1,.8f }, 0 };
    scene[2] = sphere_t{ vec3_t{ 0.95f,0,0 }, 0.05f, {.9f ,1, 1 }, 0 };
    *(disk_t*)(scene + size) = disk_t{ {0,0,0}, 0.25f, 0.6f, {1,1,1} ,{0,0,1} };

    hipMemcpyToSymbolAsync(HIP_SYMBOL(test_const), scene, sizeof(sphere_t) * size + sizeof(disk_t),0, hipMemcpyHostToDevice, stream);
    hipMemcpyToSymbolAsync(HIP_SYMBOL(cam_const), cam, sizeof(camera_t), 0, hipMemcpyHostToDevice, stream);
}

void freeScene(sphere ** scene, int count) {
    hipFree(scene);
}

void freeScene(sphere_t* scene) {
    hipFree(scene);
}


cv::cuda::GpuMat gpu_img;

cv::Mat renderScene(cv::cuda::GpuMat hdri, int img_w, int img_h, float& angle, sphere_t* scene, disk_t* disk, camera_t* cam) {
    cv::Mat3f img(img_h, img_w);
    if(gpu_img.empty())
        gpu_img.upload(img);

    dim3 grid_size(img_w / 16, img_h / 8);
    dim3 block_size(16, 8);
    camera_t* cam_gpu;

    hipMalloc(&cam_gpu, sizeof(camera_t));
    hipMemcpyAsync(cam_gpu, cam, sizeof(camera_t), hipMemcpyHostToDevice);

    
    render_base << < grid_size, block_size >> > (gpu_img, hdri, img_w, img_h, cam_gpu, scene, 2, (disk_t*)disk);
 
    gpu_img.download(img);
    hipFree(cam_gpu);
    return img;
}

cv::Mat renderScene(cv::cuda::GpuMat hdri, int img_w, int img_h, float& angle) {
    cv::Mat3f img(img_h, img_w);
    if (gpu_img.empty())
        gpu_img.upload(img);

    dim3 grid_size(img_w / 16, img_h / 8);
    dim3 block_size(16, 8);

    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(render_shared), hipFuncCachePreferEqual);
    render_shared <<< grid_size, block_size >>> (gpu_img, hdri, img_w, img_h,  2);

    gpu_img.download(img);
    return img;
}

cv::Mat renderSceneConst(cv::cuda::GpuMat hdri, int img_w, int img_h, float& angle) {
    cv::Mat3f img(img_h, img_w);
    

    if (gpu_img.empty())
        gpu_img.upload(img);

    dim3 grid_size(img_w /16,  img_h / 8);
    dim3 block_size(16, 8);

    render_constant <<< grid_size, block_size >> > (gpu_img, hdri, img_w, img_h, 2);
    gpu_img.download(img);
    return img;
}